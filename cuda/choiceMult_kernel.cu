#include "hip/hip_runtime.h"
/* Copyright (c) 2015, Julian Straub <jstraub@csail.mit.edu> Licensed
 * under the MIT license. See the license file LICENSE.
 */

#include <stdint.h>
#include <nvidia/hip/hip_runtime_api.h>

  __device__ uint wang_hash(uint seed)
  {
    seed = (seed ^ 61) ^ (seed >> 16);
    seed *= 9;
    seed = seed ^ (seed >> 4);
    seed *= 0x27d4eb2d;
    seed = seed ^ (seed >> 15);
    return seed;
  }

  __device__ uint rand_xorshift(uint rng_state)
  {
    // Xorshift algorithm from George Marsaglia's paper
    rng_state ^= (rng_state << 13);
    rng_state ^= (rng_state >> 17);
    rng_state ^= (rng_state << 5);
    return rng_state;
  }

#define N_PER_THREAD 1

  template<typename T>
  __global__ void choiceMult_kernel(T* pdfs, uint32_t *z, uint32_t N, uint32_t M,
    uint32_t seed)
  {
    //int tid = threadIdx.x;
    int idx = (threadIdx.x + blockIdx.x*blockDim.x)*N_PER_THREAD;
    if(idx <= N-N_PER_THREAD)
    {
#pragma unroll
      for(uint32_t k=0; k<N_PER_THREAD; ++k)
      {
        int idk = idx+k;
        // obtain 32 bit random int and map it into 0.0 to 1.0
        T rnd = (wang_hash(idk+seed)*2.3283064365386963e-10);
        T cdf = pdfs[idk];
        uint32_t z_i = M-1;
        for (int i=1; i<M; ++i)
        {
          if(rnd <= cdf)
          {
            z_i = i-1;
            break;
          }
          cdf += pdfs[idk+i*N];
        }
        z[idk] = z_i;
      }
    }
  }

  template<typename T>
  __global__ void choiceMultLogPdf_kernel(T* logPdfs, uint32_t *z, 
      uint32_t N, uint32_t M, uint32_t seed)
  {
    //int tid = threadIdx.x;
    int idx = (threadIdx.x + blockIdx.x*blockDim.x)*N_PER_THREAD;
    if(idx <= N-N_PER_THREAD)
    {
#pragma unroll
      for(uint32_t k=0; k<N_PER_THREAD; ++k)
      {
        int idk = idx+k;
        // obtain 32 bit random int and map it into 0.0 to 1.0
        T rnd = (wang_hash(idk+seed)*2.3283064365386963e-10);
        T cdf = exp(logPdfs[idk]);
        uint32_t z_i = M-1;
        for (int i=1; i<M; ++i)
        {
          if(rnd <= cdf)
          {
            z_i = i-1;
            break;
          }
          cdf += exp(logPdfs[idk+i*N]);
        }
        z[idk] = z_i;
      }
    }
  }

  template<typename T>
  __global__ void choiceMultLogPdfUnnormalizedGpu_kernel(T* pdfs, uint32_t *z,
    uint32_t N, uint32_t M, uint32_t seed)
  {
    //int tid = threadIdx.x;
    int idx = (threadIdx.x + blockIdx.x*blockDim.x)*N_PER_THREAD;
    if(idx <= N-N_PER_THREAD)
    {
#pragma unroll
      for(uint32_t k=0; k<N_PER_THREAD; ++k)
      {
        int idk = idx+k;
        // obtain 32 bit random int and map it into 0.0 to 1.0
        T rnd = (wang_hash(idk+seed)*2.3283064365386963e-10);
        // normalizer for logPdf
        T maxLog = -9999999.; 
        for (int k=0; k<M; ++k)
          if(maxLog < pdfs[idk + k*N]) 
            maxLog = pdfs[idk + k*N];
        T normalizer = 0;
        for (int k=0; k<M; ++k)
          normalizer += exp(pdfs[idk + k*N] - maxLog);
        normalizer = log(normalizer) + maxLog;
        
        T cdf = exp(pdfs[idk]-normalizer);
        uint32_t z_i = M-1;
        for (int i=1; i<M; ++i)
        {
        if(rnd <= cdf)
        {
          z_i = i-1;
          break;
        }
        cdf += exp(pdfs[idk+i*N]-normalizer);
      }
      z[idk] = z_i;
    }
  }
}

template<typename T>
__global__ void unif_kernel(T* u, uint32_t N, uint32_t seed)
{
  //int tid = threadIdx.x;
  int idx = (threadIdx.x + blockIdx.x*blockDim.x)*N_PER_THREAD;
  if(idx <= N-N_PER_THREAD)
  {
    // obtain 32 bit random int and map it into 0.0 to 1.0
//    u[idx] = (wang_hash(idx+seed)*2.3283064365386963e-10);
//    u[idx+N/4] = (wang_hash(idx+seed)*2.3283064365386963e-10);
//    u[idx+N/2] = (wang_hash(idx+seed)*2.3283064365386963e-10);
//    u[idx+(3*N)/4] = (wang_hash(idx+seed)*2.3283064365386963e-10);

#pragma unroll
  for(uint32_t i=0; i<N_PER_THREAD; ++i)
    u[idx+i] = (wang_hash(idx+i+seed)*2.3283064365386963e-10);
//    u[idx+1] = (wang_hash(idx+1+seed)*2.3283064365386963e-10);
//    u[idx+2] = (wang_hash(idx+2+seed)*2.3283064365386963e-10);
//    u[idx+3] = (wang_hash(idx+3+seed)*2.3283064365386963e-10);
  }
}


// assumes that pdfs are copied to device already
extern void choiceMultGpu(double* d_pdf, uint32_t* d_z, uint32_t N, uint32_t M,
  uint32_t seed)
{
  dim3 threads(256,1,1);
  dim3 blocks(N/(256*N_PER_THREAD)+(N%(256*N_PER_THREAD)>0?1:0), 1,1);
  choiceMult_kernel<double><<<blocks,threads>>>(d_pdf,d_z,N,M,seed);
  checkCudaErrors(hipDeviceSynchronize());
};
extern void choiceMultGpu(float* d_pdf, uint32_t* d_z, uint32_t N, uint32_t M,
  uint32_t seed)
{
  dim3 threads(256,1,1);
  dim3 blocks(N/(256*N_PER_THREAD)+(N%(256*N_PER_THREAD)>0?1:0), 1,1);
  choiceMult_kernel<float><<<blocks,threads>>>(d_pdf,d_z,N,M,seed);
  checkCudaErrors(hipDeviceSynchronize());
};

extern void choiceMultLogPdfGpu(double* d_logPdf, uint32_t* d_z, uint32_t N, 
    uint32_t M, uint32_t seed)
{
  dim3 threads(256,1,1);
  dim3 blocks(N/(256*N_PER_THREAD)+(N%(256*N_PER_THREAD)>0?1:0), 1,1);
  choiceMultLogPdf_kernel<double><<<blocks,threads>>>(d_logPdf,d_z,N,M,seed);
  checkCudaErrors(hipDeviceSynchronize());
};
extern void choiceMultLogPdfGpu(float* d_logPdf, uint32_t* d_z, uint32_t N, 
    uint32_t M, uint32_t seed)
{
  dim3 threads(256,1,1);
  dim3 blocks(N/(256*N_PER_THREAD)+(N%(256*N_PER_THREAD)>0?1:0), 1,1);
  choiceMultLogPdf_kernel<float><<<blocks,threads>>>(d_logPdf,d_z,N,M,seed);
  checkCudaErrors(hipDeviceSynchronize());
};

extern void choiceMultLogPdfUnNormalizedGpu(double* d_pdf, uint32_t* d_z, 
  uint32_t N, uint32_t M, uint32_t seed)
{
  dim3 threads(256,1,1);
  dim3 blocks(N/(256*N_PER_THREAD)+(N%(256*N_PER_THREAD)>0?1:0), 1,1);
  choiceMultLogPdfUnnormalizedGpu_kernel<double><<<blocks,threads>>>(d_pdf,d_z,N,
    M,seed);
  checkCudaErrors(hipDeviceSynchronize());
};
extern void choiceMultLogPdfUnNormalizedGpu(float* d_pdf, uint32_t* d_z, 
  uint32_t N, uint32_t M, uint32_t seed)
{
  dim3 threads(256,1,1);
  dim3 blocks(N/(256*N_PER_THREAD)+(N%(256*N_PER_THREAD)>0?1:0), 1,1);
  choiceMultLogPdfUnnormalizedGpu_kernel<float><<<blocks,threads>>>(d_pdf,d_z,N,
    M,seed);
  checkCudaErrors(hipDeviceSynchronize());
};

extern void unifGpu(float* d_u, uint32_t N, uint32_t seed)
{
  dim3 threads(256,1,1);
  dim3 blocks(N/(256*N_PER_THREAD)+(N%(256*N_PER_THREAD)>0?1:0), 1,1);
  unif_kernel<float><<<blocks,threads>>>(d_u,N,seed);
  checkCudaErrors(hipDeviceSynchronize());
};
extern void unifGpu(double* d_u, uint32_t N, uint32_t seed)
{
  dim3 threads(256,1,1);
  dim3 blocks(N/(256*N_PER_THREAD)+(N%(256*N_PER_THREAD)>0?1:0), 1,1);
  unif_kernel<double><<<blocks,threads>>>(d_u,N,seed);
  checkCudaErrors(hipDeviceSynchronize());
};



//TODO: fast summing up of logPdfs indicated by z
template<typename T>
__global__ void sampleLikelihood_kernel(T* logPdfs, uint32_t *z, uint32_t N, uint32_t M,
  uint32_t seed)
{
  const int tid = threadIdx.x;
  const int idx = (threadIdx.x + blockIdx.x*blockDim.x)*N_PER_THREAD;
  __shared__ T sum[256]; // TODO

  if(idx <= N-N_PER_THREAD)
  {
#pragma unroll
    for(uint32_t k=0; k<N_PER_THREAD; ++k)
    {
      int idk = idx+k;
      // obtain 32 bit random int and map it into 0.0 to 1.0
      sum[tid] += logPdfs[idk+N*z[idk]];
    }
  }
}

#define K_MAX 256
/* 
 * computes the logsumexp over chuncks of rows of d_logPdf
 * each chunck has dk cols
 * d_logPdf has N rows and K cols
 * d_logNormalizer has N rows and K/dk cols
 */
template<typename T>
__global__ void logNormalizer(T* d_logPdf, T* d_logNormalizer,
    uint32_t dk, uint32_t K, uint32_t N) 
{
//  const int tid = threadIdx.x;
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;

  for(uint32_t id=idx*N_PER_THREAD; id<min(N,(idx+1)*N_PER_THREAD); ++id)
  {
    T pdf[K_MAX]; 
    for(uint32_t k=0; k<K; ++k)
      pdf[k] = d_logPdf[id+N*k];

    uint32_t kBlock=0;
    for(uint32_t k0=0; k0<K; k0+=dk)
    {
      T maxPdf = pdf[k0];
#pragma unroll
      for(uint32_t k=k0+1; k<k0+dk; ++k)
        if(maxPdf < pdf[k]) maxPdf = pdf[k];
      T logsumexp = exp(pdf[k0]-maxPdf);
#pragma unroll
      for(uint32_t k=k0+1; k<k0+dk; ++k)
        logsumexp += exp(pdf[k]-maxPdf);
      d_logNormalizer[id + N*(kBlock++)] = - log(logsumexp)+maxPdf;
    }
//#pragma unroll
//    for(uint32_t k=0; k<K; ++k)
//      d_logPdf[id+N*k] = exp(pdf[k]-logsumexp);
  }
}

/* 
 * log normalizes over chuncks of cols of d_logPdf
 * each chunck has dk cols
 * d_logPdf has N rows and K cols
 * d_logNormalizer has N rows and K/dk cols
 */
template<typename T>
__global__ void logNormalize(T* d_logPdf,
    uint32_t dk, uint32_t K, uint32_t N) 
{
//  const int tid = threadIdx.x;
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;

  for(uint32_t id=idx*N_PER_THREAD; id<min(N,(idx+1)*N_PER_THREAD); ++id)
  {
    T pdf[K_MAX]; 
    for(uint32_t k=0; k<K; ++k)
      pdf[k] = d_logPdf[id+N*k];

    for(uint32_t k0=0; k0<K; k0+=dk)
    {
      T maxPdf = pdf[k0];
#pragma unroll
      for(uint32_t k=k0+1; k<k0+dk; ++k)
        if(maxPdf < pdf[k]) maxPdf = pdf[k];
      T logsumexp = exp(pdf[k0]-maxPdf);
#pragma unroll
      for(uint32_t k=k0+1; k<k0+dk; ++k)
        logsumexp += exp(pdf[k]-maxPdf);
      logsumexp = log(logsumexp)+maxPdf;
#pragma unroll
      for(uint32_t k=k0; k<k0+dk; ++k)
        d_logPdf[id+N*k] = pdf[k]-logsumexp;
    }
//#pragma unroll
//    for(uint32_t k=0; k<K; ++k)
//      d_logPdf[id+N*k] = exp(pdf[k]-logsumexp);
  }
}

/* add logPi + logNormalizer to logPdf for each block of dk columns
 * logPi is length K/dk
 * d_logPdf has N rows and K cols
 * d_logNormalizer has N rows and K/dk cols
 */
template<typename T>
__global__ void logAddTopLevel(T* d_logPdf, T* d_logNormalizer, T* d_logPi,
    uint32_t dk, uint32_t K, uint32_t N) 
{
//  const int tid = threadIdx.x;
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;

  for(uint32_t id=idx*N_PER_THREAD; id<min(N,(idx+1)*N_PER_THREAD); ++id)
  {
#pragma unroll
    for(uint32_t k=0; k<K; ++k)
    {
      uint32_t kBlock = k/dk;
      d_logPdf[id+ N*k] += d_logPi[kBlock] + d_logNormalizer[id + N*kBlock];
    }
//#pragma unroll
//    for(uint32_t k=0; k<K; ++k)
//      d_logPdf[id+N*k] = exp(pdf[k]-logsumexp);
  }
}

extern void logNormalizerGpu(float* d_logPdf, float* d_logNormalizer, 
    uint32_t dk, uint32_t K, uint32_t N)
{
  dim3 threads(256,1,1);
  dim3 blocks(N/(256*N_PER_THREAD)+(N%(256*N_PER_THREAD)>0?1:0), 1,1);
  logNormalizer<float><<<blocks,threads>>>(d_logPdf,d_logNormalizer,dk,K,N);
  checkCudaErrors(hipDeviceSynchronize());
};

extern void logNormalizerGpu(double* d_logPdf, double* d_logNormalizer, 
    uint32_t dk, uint32_t K, uint32_t N)
{
  dim3 threads(256,1,1);
  dim3 blocks(N/(256*N_PER_THREAD)+(N%(256*N_PER_THREAD)>0?1:0), 1,1);
  logNormalizer<double><<<blocks,threads>>>(d_logPdf,d_logNormalizer,dk,K,N);
  checkCudaErrors(hipDeviceSynchronize());
};

extern void logNormalizeGpu(float* d_logPdf,
    uint32_t dk, uint32_t K, uint32_t N)
{
  dim3 threads(256,1,1);
  dim3 blocks(N/(256*N_PER_THREAD)+(N%(256*N_PER_THREAD)>0?1:0), 1,1);
  logNormalize<float><<<blocks,threads>>>(d_logPdf,dk,K,N);
  checkCudaErrors(hipDeviceSynchronize());
};

extern void logNormalizeGpu(double* d_logPdf, 
    uint32_t dk, uint32_t K, uint32_t N)
{
  dim3 threads(256,1,1);
  dim3 blocks(N/(256*N_PER_THREAD)+(N%(256*N_PER_THREAD)>0?1:0), 1,1);
  logNormalize<double><<<blocks,threads>>>(d_logPdf,dk,K,N);
  checkCudaErrors(hipDeviceSynchronize());
};

extern void logAddTopLevelGpu(float* d_logPdf, float* d_logNormalizer, 
    float* d_logPi, uint32_t dk, uint32_t K, uint32_t N)
{
  dim3 threads(256,1,1);
  dim3 blocks(N/(256*N_PER_THREAD)+(N%(256*N_PER_THREAD)>0?1:0), 1,1);
  logAddTopLevel<float><<<blocks,threads>>>(d_logPdf,d_logNormalizer,d_logPi, 
      dk,K,N);
  checkCudaErrors(hipDeviceSynchronize());
};

extern void logAddTopLevelGpu(double* d_logPdf, double* d_logNormalizer, 
    double* d_logPi, uint32_t dk, uint32_t K, uint32_t N)
{
  dim3 threads(256,1,1);
  dim3 blocks(N/(256*N_PER_THREAD)+(N%(256*N_PER_THREAD)>0?1:0), 1,1);
  logAddTopLevel<double><<<blocks,threads>>>(d_logPdf,d_logNormalizer,d_logPi, 
      dk,K,N);
  checkCudaErrors(hipDeviceSynchronize());
};
